#include "hip/hip_runtime.h"
#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define THREADS_PER_BLOCK 512

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

int conflict_free_offset(int n)
{
	return ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS));
}

 __global__ void prescan(float* cdf, int* histogram, int n)
 {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
		cdf[index] = histogram[index] / n;
 }
 
 __global__ void scan(float *g_odata, float *g_idata, int n)
{
	extern __shared__ float temp[];  // allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	
	int ai = thid;
	int bi = thid + (n/2);
	int bankOffsetA = conflict_free_offset(ai);
	int bankOffsetB = conflict_free_offset(bi);
	temp[ai + bankOffsetA] = g_idata[ai];
	temp[bi + bankOffsetB] = g_idata[bi];
	
	for (int d = n>>1; d > 0; d >>= 1)                    // build sum in place up the tree
	{ 
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += conflict_free_offset(ai);
			bi += conflict_free_offset(bi);
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	if (thid==0)
		temp[n - 1 + conflict_free_offset(n - 1)] = 0;
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)                     
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			ai += conflict_free_offset(ai);
			bi += conflict_free_offset(bi);
			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t; 
		}
	}
	__syncthreads();
	g_odata[ai] = temp[ai + bankOffsetA];
	g_odata[bi] = temp[bi + bankOffsetB];
}

__global__ void vectorAdd(float* inputImage, unsigned char* ucharImage, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        ucharImage[index] = (unsigned char) (255 * inputImage[index]);

}

__global__ void greyScaleTransf(unsigned char* ucharImage, unsigned char* greyImage, int n) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int index = col + row * n;
  unsigned char r;
  unsigned char g;
  unsigned char b;

  if (col < n && row < n) {
    r = ucharImage[3*index];
    g = ucharImage[3*index + 1];
    b = ucharImage[3*index + 2];
    greyImage[index] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
  }
}

__global__ void histogram_comput(int* histogram, unsigned char* greyImage, int n) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < n) {
    int c = (int) greyImage[index];
    atomicAdd(&histogram[c], 1);
  }
}

int main(int argc, char **argv) {

  /* parse the input arguments */
  wbImage_t inputImage = wbImport(argv[1]);
  
  int imageWidth    = wbImage_getWidth(inputImage);
  int imageHeight   = wbImage_getHeight(inputImage);
  int imageChannels = wbImage_getChannels(inputImage);
  
  //step 1
  float *imageData = wbImage_getData(inputImage);
  float *d_imageData;
  
  unsigned char *ucharImage;
  unsigned char *d_ucharImage;
  
  int heightPerWidth = imageWidth * imageHeight;
  int max = heightPerWidth * imageChannels;
  
  int ucharImageSize = max * sizeof(unsigned char);
  int imageFloatSize = max * sizeof(float);
  
  hipMalloc((void**)&d_imageData, imageFloatSize);
  hipMalloc((void**)&d_ucharImage, ucharImageSize);
  
  ucharImage = (unsigned char *)malloc(ucharImageSize);
  
  hipMemcpy(d_imageData, imageData, imageFloatSize, hipMemcpyHostToDevice);
  
  vectorAdd<<<max/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_imageData, d_ucharImage, max);
  
  hipMemcpy(d_ucharImage, ucharImage, ucharImageSize, hipMemcpyDeviceToHost);
  
  free(imageData);
  hipFree(d_imageData);
  
  //step 2
  unsigned char *grayImage;
  unsigned char *d_grayImage;
  
  int grayImageSize = heightPerWidth * sizeof(unsigned char);
  
  hipMalloc((void**)&d_grayImage, grayImageSize);
  
  grayImage = (unsigned char *)malloc(grayImageSize);
  
  dim3 dimBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  dim3 dimGrid((int)ceil(heightPerWidth/dimBlock.x), (int)ceil(heightPerWidth/dimBlock.y));
  
  greyScaleTransf<<<dimGrid, dimBlock>>>(d_ucharImage, d_grayImage, heightPerWidth);
  
  hipMemcpy(d_grayImage, grayImage, grayImageSize, hipMemcpyDeviceToHost);
  
  //step 3
  int *histogram;
  int *d_histogram;
  
  int histogramSize = HISTOGRAM_LENGTH * sizeof(int);
  
  hipMalloc((void**)&d_histogram, histogramSize);
  
  histogram = (int *)malloc(histogramSize);
  
  cudaMemSet((void**)&d_histogram, 0, histogramSize);
  
  histogram_comput<<<heightPerWidth/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_histogram, d_grayImage, heightPerWidth);
  
  hipMemcpy(d_histogram, histogram, histogramSize, hipMemcpyDeviceToHost);
  
  //step 4
  float *cdf;
  float *d_cdf;
  
  int cdfSize = HISTOGRAM_LENGTH * sizeof(float);
  
  hipMalloc((void**)&d_cdf, cdfSize);
  
  cdf = (float *)malloc(cdfSize);
  
  prescan<<<HISTOGRAM_LENGTH,1>>>(d_cdf, d_histogram, HISTOGRAM_LENGTH);
  
  float *finalCdf;
  float *d_finalCdf;
  
  int finalCdfSize = HISTOGRAM_LENGTH * sizeof(float);
  
  hipMalloc((void**)&d_finalCdf, finalCdfSize);
  
  finalCdf = (float *)malloc(finalCdfSize);
  
  scan<<<HISTOGRAM_LENGTH,1>>>(finalCdf, d_cdf, HISTOGRAM_LENGTH);
  
  hipMemcpy(d_finalCdf, finalCdf, finalCdfSize, hipMemcpyDeviceToHost);
  
  free(ucharImage);
  free(grayImage);
  free(histogram);
  free(cdf);
  free(finalCdf);
  
  hipFree(d_ucharImage);
  hipFree(d_grayImage);
  hipFree(d_histogram);
  hipFree(d_cdf);
  hipFree(d_finalCdf);
  
 
  
  
  return 0;
}