#include "hip/hip_runtime.h"
#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define THREADS_PER_BLOCK 512



__global__ void vectorAdd(float* inputImage, unsigned char* ucharImage, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        ucharImage[index] = (unsigned char) (255 * inputImage[index]);

}




int main(int argc, char **argv) {

  /* parse the input arguments */
  wbImage_t inputImage = wbImport(argv[1]);

  int imageWidth    = wbImage_getWidth(inputImage);
  int imageHeight   = wbImage_getHeight(inputImage);
  int imageChannels = wbImage_getChannels(inputImage);

  //step 1
  float *imageData = wbImage_getData(inputImage);
  float *d_imageData;

  unsigned char *ucharImage;
  unsigned char *d_ucharImage;

  int heightPerWidth = imageWidth * imageHeight;
  int max = heightPerWidth * imageChannels;

  int ucharImageSize = max * sizeof(unsigned char);
  int imageFloatSize = max * sizeof(float);

  hipMalloc((void**)&d_imageData, imageFloatSize);
  hipMalloc((void**)&d_ucharImage, ucharImageSize);

  ucharImage = (unsigned char *)malloc(ucharImageSize);

  hipMemcpy(d_imageData, imageData, imageFloatSize, hipMemcpyHostToDevice);

  vectorAdd<<<max/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_imageData, d_ucharImage, max);

  hipMemcpy(d_ucharImage, ucharImage, ucharImageSize, hipMemcpyHostToHost);
  printf("meu, ok?");
  free(imageData);
  free(ucharImage);

  hipFree(d_imageData);
  hipFree(d_ucharImage);





  return 0;
}
