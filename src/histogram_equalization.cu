#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define THREADS_NUMBER 512


    void histogram_equalization(wbImage_t& inputImage, wbImage_t& outputImage) {

    //TODO

}


__global__ void vectorAdd(float*& inputImage, unsigned char*& ucharImage, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        ucharImage[index] = (unsigned char) (255 * inputImage[index]);

}


// __global__ void greyScaleTransf(unsigned char* ucharImage, unsigned char* greyImage, int n) {
//   int col = threadIdx.x + blockIdx.x * blockDim.x;
//   int row = threadIdx.y + blockIdx.y * blockDim.y;
//   int index = col + row * n;
//   unsigned char r;
//   unsigned char g;
//   unsigned char b;
//
//   if (col < n && row < n) {
//     r = ucharImage[3*index];
//     g = ucharImage[3*index + 1];
//     b = ucharImage[3*index + 2];
//     greyImage[index] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
//   }
// }
//
// __global__ void histogram_comput(int* histogramAux, unsigned char* greyImage, int n) {
//   int index = threadIdx.x + blockDim.x * blockIdx.x;
//   if(index < n) {
//     int c = (int) greyImage[index];
//     atomicAdd(&histogramAux[c], 1);
//   }
// }

int main(int argc, char **argv) {
  /* parse the input arguments */
  wbImage_t inputImage = wbImport(argv[1]);

  //wbImage_t *inputImage_;

  //hipMalloc((void**)&inputImage_, size);

  int imageWidth    = wbImage_getWidth(inputImage);
  int imageHeight   = wbImage_getHeight(inputImage);
  int imageChannels = wbImage_getChannels(inputImage);

  int valueHistogram = imageWidth * imageHeight;

  float * imageData = wbImage_getData(inputImage);
  float * imageData_;
  int size = sizeof(float*&);
  hipMalloc((void**)&imageData_, size);

  wbImage_t outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  // histogram_equalization(inputImage, outputImage);

  /* Cast the image from float to unsigned char */
  unsigned char *ucharImage_;
  unsigned char ucharImage;
  int size_char_image = sizeof(unsigned char&);

  unsigned char *ucharImageFinal_;
  unsigned char ucharImageFinal;

  hipMalloc((void**)&ucharImage_, size_char_image);
  hipMalloc((void**)&ucharImageFinal_, size_char_image);

  hipMemcpy(imageData_, &imageData, size, hipMemcpyHostToDevice);
  hipMemcpy(ucharImage_, &ucharImage, size_char_image, hipMemcpyHostToDevice);

  int max_ = imageWidth * imageHeight * imageChannels;
  vectorAdd<<<max_/THREADS_NUMBER,THREADS_NUMBER>>>(imageData_, &ucharImage, max_);
  //hipDeviceSynchronize();
  hipMemcpy(&ucharImageFinal, ucharImageFinal_, size_char_image, hipMemcpyDeviceToHost);

  //hipFree(grayImageFinal_);
  //hipFree(grayImage_);
  hipFree(ucharImageFinal_);
  hipFree(ucharImage_);
  hipFree(imageData_);
  //hipFree(histoLength);

  return 0;
}
