#include "hip/hip_runtime.h"
#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define THREADS_PER_BLOCK 512



__global__ void vectorAdd(float* inputImage, unsigned char* ucharImage, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        ucharImage[index] = (unsigned char) (255 * inputImage[index]);

}

__global__ void greyScaleTransf(unsigned char* ucharImage, unsigned char* greyImage, int n) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int index = col + row * n;
  unsigned char r;
  unsigned char g;
  unsigned char b;

  if (col < n && row < n) {
    r = ucharImage[3*index];
    g = ucharImage[3*index + 1];
    b = ucharImage[3*index + 2];
    greyImage[index] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
  }
}

int main(int argc, char **argv) {

  /* parse the input arguments */
  wbImage_t inputImage = wbImport(argv[1]);
  
  int imageWidth    = wbImage_getWidth(inputImage);
  int imageHeight   = wbImage_getHeight(inputImage);
  int imageChannels = wbImage_getChannels(inputImage);
  
  //step 1
  float *imageData = wbImage_getData(inputImage);
  float *d_imageData;
  
  unsigned char *ucharImage;
  unsigned char *d_ucharImage;
  
  int heightPerWidth = imageWidth * imageHeight;
  int max = heightPerWidth * imageChannels;
  
  int ucharImageSize = max * sizeof(unsigned char);
  int imageFloatSize = max * sizeof(float);
  
  hipMalloc((void**)&d_imageData, imageFloatSize);
  hipMalloc((void**)&d_ucharImage, ucharImageSize);
  
  ucharImage = (unsigned char *)malloc(ucharImageSize);
  
  hipMemcpy(d_imageData, imageData, imageFloatSize, hipMemcpyHostToDevice);
  
  vectorAdd<<<max/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_imageData, d_ucharImage, max);
  
  hipMemcpy(d_ucharImage, ucharImage, ucharImageSize, hipMemcpyHostToHost);
  
  free(imageData);
  hipFree(d_imageData);
  
  //step 2
  unsigned char *grayImage;
  unsigned char *d_grayImage;
  
  int grayImageSize = heightPerWidth * sizeof(unsigned char);
  
  hipMalloc((void**)&d_grayImage, grayImageSize);
  
  grayImage = (unsigned char *)malloc(grayImageSize);
  
  greyScaleTransf<<<heightPerWidth/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_ucharImage, d_grayImage, heightPerWidth)
  
  hipMemcpy(d_grayImage, grayImage, grayImageSize, hipMemcpyHostToHost);
  
  
  free(ucharImage);
  free(grayImage);
  
  hipFree(d_ucharImage);
  hipFree(d_grayImage);
  
  
  
  
  return 0;
}