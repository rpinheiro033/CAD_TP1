#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>

#define HISTOGRAM_LENGTH 256
#define THREADS_NUMBER 512


    void histogram_equalization(wbImage_t& inputImage, wbImage_t& outputImage) {

    //TODO

}


__global__ void vectorAdd(float* inputImage, unsigned char* ucharImage, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        ucharImage[index] = (unsigned char) (255 * inputImage[index]);

}


__global__ void greyScaleTransf(unsigned char* ucharImage, unsigned char* greyImage, int n) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int index = col + row * n;
  unsigned char r;
  unsigned char g;
  unsigned char b;

  if (col < n && row < n) {
    r = ucharImage[3*index];
    g = ucharImage[3*index + 1];
    b = ucharImage[3*index + 2];
    greyImage[index] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
  }
}

__global__ void histogram_comput(int* histogramAux, unsigned char* greyImage, int n) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < n) {
    int c = (int) greyImage[index];
    atomicAdd(&histogramAux[c], 1);
  }
}

int main(int argc, char **argv) {
  /* parse the input arguments */
  wbImage_t inputImage = wbImport(argv[1]);

  //wbImage_t *inputImage_;

  //hipMalloc((void**)&inputImage_, size);

  int imageWidth    = wbImage_getWidth(inputImage);
  int imageHeight   = wbImage_getHeight(inputImage);
  int imageChannels = wbImage_getChannels(inputImage);

  int valueHistogram = imageWidth * imageHeight;

  float * imageData = wbImage_getData(inputImage);
  float * imageData_;
  int size = sizeof(float*);
  hipMalloc((void**)&imageData_, size);

  wbImage_t outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  // histogram_equalization(inputImage, outputImage);

  /* Cast the image from float to unsigned char */
  unsigned char *ucharImage_;
  unsigned char ucharImage;
  int size_char_image = sizeof(unsigned char&);

  unsigned char *ucharImageFinal_;
  unsigned char ucharImageFinal;

  hipMalloc((void**)&ucharImage_, size_char_image);
  hipMalloc((void**)&ucharImageFinal_, size_char_image);

  hipMemcpy(imageData_, &imageData, size, hipMemcpyHostToDevice);
  hipMemcpy(ucharImage_, &ucharImage, size_char_image, hipMemcpyHostToDevice);

  int max_ = imageWidth * imageHeight * imageChannels;
  vectorAdd<<<max_/THREADS_NUMBER,THREADS_NUMBER>>>(imageData_, &ucharImage, max_);
  hipDeviceSynchronize();
  hipMemcpy(&ucharImageFinal, ucharImageFinal_, size_char_image, hipMemcpyDeviceToHost);

 /* Step 2 */
 //wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
 unsigned char *grayImage_;
 unsigned char grayImage;

 int size_grayImage = sizeof(unsigned char&);

 unsigned char *grayImageFinal_;
 unsigned char grayImageFinal;

 hipMalloc((void**)&grayImage_, size_grayImage);
 hipMalloc((void**)&grayImageFinal_, size_grayImage);

  hipMemcpy(grayImage_, &grayImage, size_grayImage, hipMemcpyHostToDevice);

  dim3 dimBlock(THREADS_NUMBER, THREADS_NUMBER);
  dim3 dimGrid((int)ceil(valueHistogram/dimBlock.x), (int)ceil(valueHistogram/dimBlock.y));
  greyScaleTransf<<<dimGrid, dimBlock>>>(&ucharImage, &grayImage, valueHistogram);
  hipDeviceSynchronize();
  hipMemcpy(grayImageFinal_, &grayImageFinal, size_grayImage, hipMemcpyDeviceToHost);
 //wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  int * histoLength;
  int length = sizeof(int&) * valueHistogram;

  hipMalloc((void**) &histoLength, length);
  hipMemset((void**) &histoLength, 0, length);

  for(int i = 0; i < sizeof(histoLength); i++) {
    printf("Positions of histogram: %d\n", histoLength[i]);
  }


  int * histoLengthToPrint = new int[valueHistogram];
  histogram_comput<<<valueHistogram/THREADS_NUMBER, THREADS_NUMBER>>>(histoLength, grayImageFinal_, valueHistogram);
  hipDeviceSynchronize();
  hipMemcpy(histoLength, &valueHistogram, length, hipMemcpyDeviceToHost);

  hipFree(grayImageFinal_);
  hipFree(grayImage_);
  hipFree(ucharImageFinal_);
  hipFree(ucharImage_);
  hipFree(imageData_);
  hipFree(histoLength);

  printf("imageWidth = %d\n", imageWidth);
  printf("MAX = %d\n", max_);

  return 0;
}
